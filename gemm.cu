#include <cassert>
#include <hip/hip_runtime.h>

#define ERROR(message, ...) do { fprintf(stderr, message, ##__VA_ARGS__); abort(); } while(0)

#define WARP_SIZE 32

template<const uint BN, const uint BM, const uint BK,
		 const uint WN, const uint WM, const uint WMITER,
         const uint TN, const uint TM, const uint NUM_THREADS>
__global__ void __launch_bounds__(NUM_THREADS)
	kernel_gemm(uint N, uint M, uint K, 
				const float *A, const float *B, float *C) 
{
	const uint blrow = blockIdx.y;
	const uint blcol = blockIdx.x; 

	const uint iArow = threadIdx.x / (BK / 4);
	const uint iAcol = threadIdx.x % (BK / 4);
	constexpr uint strideA = NUM_THREADS / (BK / 4);
	
	const uint iBrow = threadIdx.x / (BM / 4);
	const uint iBcol = threadIdx.x % (BM / 4);
	constexpr uint strideB = NUM_THREADS / (BM / 4);

	const uint widx = threadIdx.x / WARP_SIZE;
	const uint wrow = widx / (BM / WM);
	const uint wcol = widx % (BM / WM);

	constexpr uint WNITER = (WN*WM) / (WARP_SIZE*TN*TM*WMITER);
	constexpr uint SUBWN = WN / WNITER;
	constexpr uint SUBWM = WM / WMITER;

	const uint tidx = threadIdx.x % WARP_SIZE;
	const uint trow = tidx / (SUBWM / TM);
	const uint tcol = tidx % (SUBWM / TM);

	__shared__ float SA[BN*BK], SB[BK*BM];

	A += (blrow * BN)*K;
	B += (blcol * BM)*1;
	C += (blrow * BN + wrow * WN)*M + (blcol * BM + wcol *WM)*1;
	
	float rs[WNITER*WMITER*TN*TM] = {0}; // [WNITER][WMITER][TN][TM]
	float regA[WNITER*TN], regB[WMITER*TM];

	for (uint blit = 0; blit < (K/BK); blit++) {
		// Loading A and B to shared memory
		for (uint offA = 0; offA < BN; offA += strideA) {
			float4 tmp = reinterpret_cast<const float4 *>(&A[(iArow+offA)*K + (iAcol)*4])[0];
			// Transposing SA
			SA[(iAcol * 4 + 0)*BN +(iArow+offA)] = tmp.x;
			SA[(iAcol * 4 + 1)*BN +(iArow+offA)] = tmp.y;
			SA[(iAcol * 4 + 2)*BN +(iArow+offA)] = tmp.z;
			SA[(iAcol * 4 + 3)*BN +(iArow+offA)] = tmp.w;
		}
		for (uint offB = 0; offB < BK; offB += strideB) {
			float4 tmp = reinterpret_cast<const float4 *>(&B[(iBrow+offB)*M + iBcol*4])[0];
			SB[(iBrow+offB)*BM + iBcol*4 + 0] = tmp.x;
			SB[(iBrow+offB)*BM + iBcol*4 + 1] = tmp.y;
			SB[(iBrow+offB)*BM + iBcol*4 + 2] = tmp.z;
			SB[(iBrow+offB)*BM + iBcol*4 + 3] = tmp.w;
		}
		__syncthreads();

		// Doing computation
		for (uint k = 0; k < BK; k++) {
			for (uint iSubN = 0; iSubN < WNITER; iSubN++) {
				for (uint rn = 0; rn < TN; rn++) {
					regA[iSubN*TN + rn] = SA[k*BN + (wrow*WN + iSubN*SUBWN + trow*TN + rn)];
				}
			}
			for (uint iSubM = 0; iSubM < WMITER; iSubM++) {
				for (uint rm = 0; rm < TM; rm++) {
					regB[iSubM*TM + rm] = SB[k*BM + (wcol*WM + iSubM*SUBWM + tcol*TM + rm)];
				}
			}

			for (uint iSubN = 0; iSubN < WNITER; iSubN++) {
				for (uint iSubM = 0; iSubM < WMITER; iSubM++) {
					for (uint rn = 0; rn < TN; rn++) {
						for (uint rm = 0; rm < TM; rm++) {
							rs[iSubN*WMITER*TN*TM + iSubM*TN*TM + rn*TM + rm] += 
								regA[iSubN*TN + rn] * regB[iSubM*TM + rm];
						}
					}
				}
			}
		}
		A += BK;
		B += BK*M;
		__syncthreads();
	}
	for (uint iSubN = 0; iSubN < WNITER; iSubN++) {
		for (uint iSubM = 0; iSubM < WMITER; iSubM++) {
			for (uint rn = 0; rn < TN; rn++) {
				for (uint rm = 0; rm < TM; rm += 4) {
					float4 tmp = reinterpret_cast<float4 *>(&rs[
						iSubN*WMITER*TN*TM + iSubM*TN*TM + rn*TM + rm]
					)[0];
					reinterpret_cast<float4 *>
						(&C[(iSubN*SUBWN + trow*TN + rn)*M + (iSubM*SUBWM + tcol*TM + rm)])[0] = tmp;
				}
			}
		}
	}
}

#define CEIL(a, b) (((a)+(b)-1)/ (b))

void gemm(uint N, uint M, uint K, const float *A, const float *B, float *C) 
{
	constexpr uint NUM_THREADS = 128;
	constexpr uint BN = 128;
	constexpr uint BM = 64;
	constexpr uint BK = 16;
	constexpr uint WN = 64;
	constexpr uint WM = 32;
	constexpr uint WMITER = 1;
	constexpr uint TN = 4;
	constexpr uint TM = 4;

	static_assert(BM % TM == 0);
	static_assert(BN % TM == 0);
	static_assert(NUM_THREADS % BK == 0);
	static_assert(NUM_THREADS % BM == 0);

	assert((N % BN) == 0 && "N must be a multiple of BN\n");
	assert((M % BM) == 0 && "M must be a multiple of BM\n");
	assert((K % BK) == 0 && "K must be a multiple of BK\n");

	dim3 blockDim(NUM_THREADS);
	dim3 gridDim(M/BM, N/BN);
	kernel_gemm
		<BN, BM, BK, 
		WN, WM, WMITER,
		TN, TM, NUM_THREADS>
		<<<gridDim, blockDim>>>(N, M, K, A, B, C);
	hipError_t err = hipGetLastError();

	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}
}
