#include "hip/hip_runtime.h"
#include <stdio.h>
#include <chrono>

#include "gemm.cu"

#define TEMP_PATH "/tmp/matmul_"
#define TEST_ASSERT(expr, message, ...) do  { \
if (!(expr)) { \
	fprintf(stderr, message, ##__VA_ARGS__); \
	return false; \
} \
} while(0)

#define LOAD_VAR(x)   TEST_ASSERT(sizeof(x) == fread(&x, 1, sizeof(x), f), "Test bin format is wrong\n");
#define LOAD_ARRAY(x) TEST_ASSERT(sizeof(x) == fread(x, 1, sizeof(x), f), "Test bin format is wrong\n");
#define LOAD_PTR(x, sz) TEST_ASSERT((sz) == fread(x, 1, (sz), f), "Test bin format is wrong\n");


#define TEST_COPY_ARRAY(x, x_exp) hipMemcpy(x, x_exp, sizeof(x_exp), hipMemcpyHostToDevice)
#define TEST_COPY_PTR(x, x_exp, sz) hipMemcpy(x, x_exp, (sz), hipMemcpyHostToDevice)

#define CLOSE_EPS 1e-3

char *gpu_alloc(size_t n) 
{
	char *ptr; hipMalloc(&ptr, n);
	return ptr;
}

float *gpu_alloc_float(size_t n) 
{
	return (float*)gpu_alloc(n * sizeof(float));
}

char *cpu_alloc(size_t n) 
{
	return (char*) malloc(n);
}

float *cpu_alloc_float(size_t n) 
{
	return (float*)cpu_alloc(n * sizeof(float));
}

bool assert_close(float *gpu_data, float *exp, int n, float eps = CLOSE_EPS)
{
	float *data = cpu_alloc_float(n);
	hipMemcpy(data, gpu_data, sizeof(float) * n, hipMemcpyDeviceToHost);
	for (int i = 0; i < n; i++) {
		float diff = abs(data[i] - exp[i]);
		TEST_ASSERT(diff < eps, "Number are not close (i, diff) = (%d, %f)", i, diff);
	}
	return true;
}

bool show_gpu_data(float *gpu_data, int n) 
{
	float *data = cpu_alloc_float(n);
	hipMemcpy(data, gpu_data, sizeof(float) * n, hipMemcpyDeviceToHost);
	for (int i = 0; i < n; i++) {
		printf("%.4f ", data[i]);
	}
	printf("\n");
	return true;
}

bool show_data(float *data, int n) 
{
	for (int i = 0; i < n; i++) {
		printf("%.4f ", data[i]);
	}
	printf("\n");
	return true;
}

bool test_matmul()
{
	FILE *f = fopen(TEMP_PATH"matmul.bin", "rb");
	TEST_ASSERT(f != NULL, "Could not open test_matmul bin\n");

	int N, K, M;
	LOAD_VAR(N); LOAD_VAR(K); LOAD_VAR(M);

	float *x_exp = cpu_alloc_float(N*K);   LOAD_PTR(x_exp, N*K*sizeof(float));
	float *w_exp = cpu_alloc_float(K*M);   LOAD_PTR(w_exp, K*M*sizeof(float));
	float *out_exp = cpu_alloc_float(N*M); LOAD_PTR(out_exp, N*M*sizeof(float));
	fclose(f);

	float *x   = (float *) gpu_alloc(N*K*sizeof(float)); 
	float *w   = (float *) gpu_alloc(K*M*sizeof(float)); 
	float *out = (float *) gpu_alloc(N*M*sizeof(float)); 

	TEST_COPY_PTR(x, x_exp, N*K*sizeof(float));
	TEST_COPY_PTR(w, w_exp, K*M*sizeof(float));
	hipDeviceSynchronize();

	gemm(N, M, K, x, w, out);
	hipDeviceSynchronize();
	if (!assert_close(out, out_exp, N*M)) return false;

	for (int z = 0; z < 1000; z++) {
		auto start = std::chrono::steady_clock::now();
		gemm(N, M, K, x, w, out);
		hipDeviceSynchronize();
		auto stop = std::chrono::steady_clock::now();

		auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
		long double duration_nano = duration.count();
		double gflop = 2*(double)N*K*M;
		double gflops = gflop / duration_nano;
		printf("Foward Attention Gflops = %lf\n", gflops);
	}
	return true;
}

int main()
{
	int errors = 0;
	errors += !test_matmul();

	if (errors > 0) {
		fprintf(stderr, "Tests failed with %d errors\n", errors);
		return 1;
	}

	fprintf(stdout, "SUCESS\n");

	return 0;
}
